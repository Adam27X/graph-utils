#include "util_device.cuh"

//Note: Times are returned in seconds
void start_clock(hipEvent_t &start, hipEvent_t &end)
{
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&end));
	checkCudaErrors(hipEventRecord(start,0));
}

float end_clock(hipEvent_t &start, hipEvent_t &end)
{
	float time;
	checkCudaErrors(hipEventRecord(end,0));
	checkCudaErrors(hipEventSynchronize(end));
	checkCudaErrors(hipEventElapsedTime(&time,start,end));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(end));

	return time/(float)1000;
}

void choose_device(program_options &op)
{
	int count;
	checkCudaErrors(hipGetDeviceCount(&count));
	hipDeviceProp_t prop;

	if(op.device == -1)
	{
		//Choose the device with the greatest memory BW
		double maxMemBandwidth = 0;
		int bestdev = 0;
		for(int i=0; i<count; i++)
		{
			checkCudaErrors(hipGetDeviceProperties(&prop,i));
			double memBandwidth = (prop.memoryClockRate * 1000.0) * (prop.memoryBusWidth / 8 * 2) / 1.0e9;
			if(memBandwidth > maxMemBandwidth)
			{
				maxMemBandwidth = memBandwidth;
				bestdev = i;
			}
		}
		checkCudaErrors(hipSetDevice(bestdev));
		checkCudaErrors(hipGetDeviceProperties(&prop,bestdev));
	}
	else if((op.device < -1) || (op.device >= count))
	{
		std::cerr << "Invalid device argument. Valid devices on this machine range from 0 to " << count-1 << std::endl;
		exit(-1);
	}
	else
	{
		checkCudaErrors(hipSetDevice(op.device));
		checkCudaErrors(hipGetDeviceProperties(&prop,op.device));
	}

	size_t free_mem, total_mem;
	checkCudaErrors(hipMemGetInfo(&free_mem,&total_mem));
	double memBandwidth = (prop.memoryClockRate * 1000.0) * (prop.memoryBusWidth / 8 * 2) / 1.0e9;
	int runtime_version;
	checkCudaErrors(hipRuntimeGetVersion(&runtime_version));
	std::string dev_name(prop.name);
	if(dev_name.find("Tesla") != std::string::npos)
	{
		op.isTesla = true;
	}

	std::cout << "CUDA Runtime Version: " << runtime_version << std::endl;
	std::cout << "Chosen Device: " << prop.name << std::endl;
	std::cout << "Compute Capability: " << prop.major << "." << prop.minor << std::endl;
	std::cout << "Number of Streaming Multiprocessors: " << prop.multiProcessorCount << std::endl;
	std::cout << "Size of Global Memory: " << (total_mem/(double)(1 << 30)) << " GB" << std::endl;
	std::cout << "Memory Bandwidth: " << memBandwidth << " GB/s " << std::endl;
}

/*void start_power_sample(program_options op, pthread_t &thread, long period)
{
        if(op.device != 0) //Could add an isTelsa flag, or use the NVML library directly to ensure that power can be measured from the GPU of interest
        {
                std::cerr << "Warning: Power can only be measured for Tesla GPUs." << std::endl;
        }
        else
        {
                //Spawn pthread for power measurement
                psample = new bool;
                *psample = true;
                pthread_create(&thread, NULL, power_sample, (void*)period);
                //std::cout << "Thread created." << std::endl;
                //std::cout << "Psample in main: " << *psample << std::endl;
        }
}*/

