#include "betweenness_centrality.cuh"

//TODO: Return reference
std::vector< std::vector<float> > betweenness_centrality_setup(const device_graph &g, int start, int end)
{
	//For now, use "standard" grid/block sizes. These can be tuned later on.
	dim3 dimGrid, dimBlock;
        //Returns number of source vertices to store for verification purposes
        size_t sources_to_store = configure_grid(dimGrid,dimBlock,start,end);

	//Device pointers
	int *d_d, *Q_d, *Q2_d, *S_d, *endpoints_d;
	unsigned long long *sigma_d;
	float *delta_d;
	pitch p;
	hipEvent_t start_event, end_event;

	//Allocate algorithm-specific memory
	start_clock(start_event,end_event);
	checkCudaErrors(hipMallocPitch((void**)&d_d,&p.d,sizeof(int)*g.n,sources_to_store));
	checkCudaErrors(hipMallocPitch((void**)&sigma_d,&p.sigma,sizeof(unsigned long long)*g.n,sources_to_store));
	checkCudaErrors(hipMallocPitch((void**)&delta_d,&p.delta,sizeof(float)*g.n,sources_to_store));
	checkCudaErrors(hipMallocPitch((void**)&Q_d,&p.Q,sizeof(int)*g.n,dimGrid.x));
	checkCudaErrors(hipMallocPitch((void**)&Q2_d,&p.Q2,sizeof(int)*g.n,dimGrid.x));
	checkCudaErrors(hipMallocPitch((void**)&S_d,&p.S,sizeof(int)*g.n,dimGrid.x));
	checkCudaErrors(hipMallocPitch((void**)&endpoints_d,&p.endpoints,sizeof(int)*g.n,dimGrid.x));
	thrust::device_vector<float> bc_d(g.n,0);

        size_t GPU_memory_requirement = sizeof(int)*g.n*sources_to_store + 4*sizeof(int)*g.n*dimGrid.x + sizeof(int)*(g.n+1) + sizeof(int)*(g.m) + sizeof(unsigned long long)*g.n*sources_to_store + sizeof(float)*g.n*sources_to_store + sizeof(float)*g.n;
        std::cout << "BC memory requirement: " << GPU_memory_requirement/(1 << 20) << " MB" << std::endl;

	betweenness_centrality<<<dimGrid,1024>>>(thrust::raw_pointer_cast(g.R.data()),thrust::raw_pointer_cast(g.C.data()),g.n,d_d,sigma_d,delta_d,thrust::raw_pointer_cast(bc_d.data()),Q_d,Q2_d,S_d,endpoints_d,p,start,end);
	checkCudaErrors(hipPeekAtLastError());

        std::vector< std::vector<float> > delta_h;
        transfer_result(g,delta_d,p.delta,sources_to_store,delta_h);

	//Free algorithm-specific memory
	checkCudaErrors(hipFree(endpoints_d));
	checkCudaErrors(hipFree(S_d));
	checkCudaErrors(hipFree(Q2_d));
	checkCudaErrors(hipFree(Q_d));
	checkCudaErrors(hipFree(delta_d));
	checkCudaErrors(hipFree(sigma_d));
	checkCudaErrors(hipFree(d_d));
	float time = end_clock(start_event,end_event);

	std::cout << "Time for shuffle-based BC: " << std::setprecision(9) << time << " s" << std::endl;

	return delta_h;
}

