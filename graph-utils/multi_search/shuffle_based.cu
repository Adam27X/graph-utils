#include "hip/hip_runtime.h"
#include "shuffle_based.cuh"
#include "common.cuh"

std::vector< std::vector<int> > multi_search_shuffle_based_setup(const device_graph &g, int start, int end)
{
	//For now, use "standard" grid/block sizes. These can be tuned later on.
	dim3 dimGrid, dimBlock;
        //Returns number of source vertices to store for verification purposes
        size_t sources_to_store = configure_grid(dimGrid,dimBlock,start,end);

	//Device pointers
	int *d_d, *Q_d, *Q2_d;
	size_t pitch_d, pitch_Q, pitch_Q2;
	hipEvent_t start_event, end_event;

	//Allocate algorithm-specific memory
	start_clock(start_event,end_event);
	checkCudaErrors(hipMallocPitch((void**)&d_d,&pitch_d,sizeof(int)*g.n,sources_to_store));
	checkCudaErrors(hipMallocPitch((void**)&Q_d,&pitch_Q,sizeof(int)*g.n,dimGrid.x));
	checkCudaErrors(hipMallocPitch((void**)&Q2_d,&pitch_Q2,sizeof(int)*g.n,dimGrid.x));

        size_t GPU_memory_requirement = sizeof(int)*g.n*sources_to_store + 2*sizeof(int)*g.n*dimGrid.x + sizeof(int)*(g.n+1) + sizeof(int)*(g.m);
        std::cout << "Shuffle based memory requirement: " << GPU_memory_requirement/(1 << 20) << " MB" << std::endl;

	multi_search_shuffle_based<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(g.R.data()),thrust::raw_pointer_cast(g.C.data()),g.n,d_d,pitch_d,Q_d,pitch_Q,Q2_d,pitch_Q2,start,end);
	checkCudaErrors(hipPeekAtLastError());

        std::vector< std::vector<int> > d_host_vector;
        transfer_result(g,d_d,pitch_d,sources_to_store,d_host_vector);

	//Free algorithm-specific memory
	checkCudaErrors(hipFree(Q2_d));
	checkCudaErrors(hipFree(Q_d));
	checkCudaErrors(hipFree(d_d));
	float time = end_clock(start_event,end_event);

	std::cout << "Time for shuffle-based neighbor gathering: " << std::setprecision(9) << time << " s" << std::endl;

	return d_host_vector;
}

//Wrappers
__global__ void multi_search_shuffle_based(const int *R, const int *C, const int n, int *d, size_t pitch_d, int *Q, size_t pitch_Q, int *Q2, size_t pitch_Q2, const int start, const int end)
{
        auto null_lamb_1 = [](int){};
	auto null_lamb_2 = [](int,int){};
	auto null_lamb_3 = [](int*,int,int){};
        multi_search(R,C,n,d,pitch_d,Q,pitch_Q,Q2,pitch_Q2,start,end,null_lamb_1,null_lamb_2,null_lamb_3);
}

__global__ void diameter_sampling(const int *R, const int *C, const int n, int *d, size_t pitch_d, int *Q, size_t pitch_Q, int *Q2, size_t pitch_Q2, int *max, const int start, const int end)
{
        auto max_lamb = [max](int v) //Using a separate variable for kinder syntax highlighting in vim
        {
                if(v != INT_MAX)
                {
                        atomicMax(max,v);
                }
        };

	auto null_lamb_1 = [](int,int){};
	auto null_lamb_2 = [](int*,int,int){};

        multi_search(R,C,n,d,pitch_d,Q,pitch_Q,Q2,pitch_Q2,start,end,max_lamb,null_lamb_1,null_lamb_2);
}

__global__ void all_pairs_shortest_paths(const int *R, const int *C, const int n, int *d, size_t pitch_d, unsigned long long *sigma, size_t pitch_sigma, int *Q, size_t pitch_Q, int *Q2, size_t pitch_Q2, const int start, const int end)
{
	auto null_lamb = [](int){};

	auto init_sigma_row = [sigma,pitch_sigma] (int k, int i)
	{
		unsigned long long *sigma_row = (unsigned long long*)((char*)sigma + blockIdx.x*pitch_sigma);
		if(k == i)
		{
			sigma_row[k] = 1;
		}
		else
		{
			sigma_row[k] = 0;
		}
	};

	auto update_sigma_row = [sigma,pitch_sigma] (int *d_row, int v, int w)
	{
		if(d_row[w] == d_row[v]+1)
		{
			unsigned long long *sigma_row = (unsigned long long*)((char*)sigma + blockIdx.x*pitch_sigma);
			atomicAdd(&sigma_row[w],sigma_row[v]);
		}
	};

	multi_search(R,C,n,d,pitch_d,Q,pitch_Q,Q2,pitch_Q2,start,end,null_lamb,init_sigma_row,update_sigma_row);
}
