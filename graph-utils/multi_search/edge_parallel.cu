#include "hip/hip_runtime.h"
#include "edge_parallel.cuh"

std::vector< std::vector<int> > multi_search_edge_parallel_setup(device_graph &g, int start, int end)
{
	//For now, use "standard" grid/block sizes. These can be tuned later on.
	dim3 dimGrid, dimBlock;
	dimGrid.x = 14;
	dimGrid.y = 1;
	dimGrid.z = 1;

	dimBlock.x = 1024;
	dimBlock.y = 1;
	dimBlock.z = 1;

	//Device pointers
	int *d_d;
	size_t pitch_d;
	hipEvent_t start_event, end_event;

	//Allocate algorithm-specific memory
	start_clock(start_event,end_event);
	checkCudaErrors(hipMallocPitch((void**)&d_d,&pitch_d,sizeof(int)*g.n,end-start));

	multi_search_edge_parallel<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(g.F.data()),thrust::raw_pointer_cast(g.C.data()),g.n,g.m,d_d,pitch_d,start,end);
	checkCudaErrors(hipPeekAtLastError());

	//Transfer result to host. Use CUDA library calls to copy into a C-style array and then move that to a vector for convenience.
	int *d_host_array = new int[g.n*(end-start)];
	checkCudaErrors(hipMemcpy2D(d_host_array,sizeof(int)*g.n,d_d,pitch_d,sizeof(int)*g.n,(end-start),hipMemcpyDeviceToHost));
	std::vector< std::vector<int> > d_host_vector(end-start);
	for(int i=start; i<end; i++)
	{
		d_host_vector[i-start].resize(g.n);
		for(int j=0; j<g.n; j++)
		{
			d_host_vector[i-start][j] = d_host_array[i*g.n + j];
		}
	}
	delete[] d_host_array;

	//Free algorithm-specific memory
	checkCudaErrors(hipFree(d_d));
	float time = end_clock(start_event,end_event);

	std::cout << "Time for baseline edge-parallel algorithm: " << time << " s" << std::endl;

	return d_host_vector;
}

__global__ void multi_search_edge_parallel(const int *F, const int *C, const int n, const int m, int *d, size_t pitch_d, const int start, const int end)
{
	int j = threadIdx.x;

	for(int i=blockIdx.x+start; i<end; i+=gridDim.x)
	{
		//Initialization
		int *d_row = (int *)((char *)d + (i-start)*pitch_d); 
		for(int k=threadIdx.x; k<n; k+=blockDim.x)
		{
			if(k == i)
			{
				d_row[k] = 0;
			}
			else
			{
				d_row[k] = INT_MAX;
			}
		}
		__syncthreads();

		
		__shared__ bool done;
		__shared__ int current_depth;
		if(j == 0)
		{
			done = false;
			current_depth = -1;
		}
		__syncthreads();

		while(!done)
		{
			if(j == 0)
			{
				done = true;
				current_depth++;
			}
			__syncthreads();
			for(int k=threadIdx.x; k<m; k+=blockDim.x) //m undirected edges = 2m directed edges. Might want to change this nomenclature.
			{
				int v = F[k];
				if(d_row[v] == current_depth)
				{
					int w = C[k];
					if(atomicCAS(&d_row[w],INT_MAX,d_row[v]+1) == INT_MAX)
					{
						done = false;
					}
				}
			}
			__syncthreads();
		}
	}
}
