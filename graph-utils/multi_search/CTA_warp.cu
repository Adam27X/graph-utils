#include "hip/hip_runtime.h"
#include "CTA_warp.cuh"
#include "common.cuh"

std::vector< std::vector<int> > multi_search_CTA_warp_based_setup(const device_graph &g, int start, int end)
{
	//For now, use "standard" grid/block sizes. These can be tuned later on.
	dim3 dimGrid, dimBlock;
        //Returns number of source vertices to store for verification purposes
        size_t sources_to_store = configure_grid(dimGrid,dimBlock,start,end);

	//Device pointers
	int *d_d, *Q_d, *Q2_d;
	size_t pitch_d, pitch_Q, pitch_Q2;
	hipEvent_t start_event, end_event;

	//Allocate algorithm-specific memory
	start_clock(start_event,end_event);
	checkCudaErrors(hipMallocPitch((void**)&d_d,&pitch_d,sizeof(int)*g.n,sources_to_store));
	checkCudaErrors(hipMallocPitch((void**)&Q_d,&pitch_Q,sizeof(int)*g.n,dimGrid.x));
	checkCudaErrors(hipMallocPitch((void**)&Q2_d,&pitch_Q2,sizeof(int)*g.n,dimGrid.x));

        size_t GPU_memory_requirement = sizeof(int)*g.n*sources_to_store + 2*sizeof(int)*g.n*dimGrid.x + sizeof(int)*(g.n+1) + sizeof(int)*(g.m);
        std::cout << "CTA+warp memory requirement: " << GPU_memory_requirement/(1 << 20) << " MB" << std::endl;

	multi_search_CTA_warp_based<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(g.R.data()),thrust::raw_pointer_cast(g.C.data()),g.n,d_d,pitch_d,Q_d,pitch_Q,Q2_d,pitch_Q2,start,end);
	checkCudaErrors(hipPeekAtLastError());

        std::vector< std::vector<int> > d_host_vector;
        transfer_result(g,d_d,pitch_d,sources_to_store,d_host_vector);

	//Free algorithm-specific memory
	checkCudaErrors(hipFree(Q2_d));
	checkCudaErrors(hipFree(Q_d));
	checkCudaErrors(hipFree(d_d));
	float time = end_clock(start_event,end_event);

	std::cout << "Time for CTA+warp neighbor gathering: " << std::setprecision(9) << time << " s" << std::endl;

	return d_host_vector;
}

__global__ void multi_search_CTA_warp_based(const int *R, const int *C, const int n, int *d, size_t pitch_d, int *Q, size_t pitch_Q, int *Q2, size_t pitch_Q2, const int start, const int end)
{
	const int degree_threshold = 2048;
	int j = threadIdx.x;
	int warp_id = threadIdx.x/32;
	int lane_id = threadIdx.x & 0x1f;
	__shared__ int  *Q_row;
	__shared__ int *Q2_row;

	if(j == 0)
	{
		Q_row = (int*)((char*)Q + blockIdx.x*pitch_Q);
		Q2_row = (int*)((char*)Q2 + blockIdx.x*pitch_Q2);
	}
	__syncthreads();

	for(int i=blockIdx.x+start; i<end; i+=gridDim.x)
	{
		int *d_row = (int*)((char*)d + blockIdx.x*pitch_d);
		for(int k=threadIdx.x; k<n; k+=blockDim.x)
		{
			if(k == i)
			{
				d_row[k] = 0;
			}
			else
			{
				d_row[k] = INT_MAX;
			}
		}
		__syncthreads();

		__shared__ int Q_len;
		__shared__ int Q2_len;

		if(j == 0)
		{
			Q_row[0] = i;
			Q_len = 1;
			Q2_len = 0;
		}
		__syncthreads();

		while(1)
		{
			volatile __shared__ int comm[32][4]; //32 is the number of warps
			__shared__ int CTA_comm[4];
			int v, r, r_end;	
			int k = threadIdx.x;

			if(k < Q_len)
			{	
				v = Q_row[k];
				r = R[v];
				r_end = R[v+1];
			}
			else
			{
				v = -1;
				r = 0;
				r_end = 0;
			}

			//If a thread's adjlist is really large, have the entire CTA process it. 
			while(1)
			{
				__shared__ bool high_degree;
				if(threadIdx.x == 0)
				{
					high_degree = false;
				}
				__syncthreads();
				for(int m=threadIdx.x; m<Q_len; m+=blockDim.x)
				{
					if(r_end-r > degree_threshold)
					{
						high_degree = true;
						CTA_comm[0] = threadIdx.x;
					}
				}
				__syncthreads();
				while(high_degree) 
				{
					if(CTA_comm[0] == threadIdx.x)
					{
						CTA_comm[1] = r;
						CTA_comm[2] = r_end;
						CTA_comm[3] = v;
						r = 0; //Same thread cannot win twice
						r_end = 0; 
					}
					__syncthreads();
					
					int r_gather = CTA_comm[1] + threadIdx.x;
					int r_gather_end = CTA_comm[2];
					int v_new = CTA_comm[3];
					while(r_gather < r_gather_end)
					{
						volatile int w = C[r_gather];
						//Assuming no duplicate/self-edges in the graph, no atomics needed
						if(d_row[w] == INT_MAX)
						{
							d_row[w] = d_row[v_new]+1;
							int t = atomicAdd(&Q2_len,1);
							Q2_row[t] = w;
						}
						r_gather += blockDim.x;
					}
					__syncthreads();
					
					//See if another loop iteration is needed. This would be cleaner in a device function
					if(threadIdx.x == 0)
					{
						high_degree = false;
					}
					__syncthreads();
					for(int m=threadIdx.x; m<Q_len; m+=blockDim.x)
					{
						if(r_end-r > degree_threshold)
						{
							high_degree = true;
							CTA_comm[0] = threadIdx.x;
						}
					}
					__syncthreads();
				}
				k+=blockDim.x;
				if(k < Q_len)
				{
					v = Q_row[k];
					r = R[v];
					r_end = R[v+1];
				}
				else
				{
					v = -1;
					r = 0;
					r_end = 0;
				}

				if((k-threadIdx.x) >= Q_len) //If thread 0 doesn't have work, the entire block is done
				{
					break;
				}
			}

			//Next, use the warp-based approach
			k = threadIdx.x; //Go back to the beginning of the queue

			if(k < Q_len)
			{	
				v = Q_row[k];
				r = R[v];
				r_end = R[v+1];
				if(r_end-r > degree_threshold) //These were already taken care of by the entire CTA
				{
					r = 0;
					r_end = 0;
				}
			}
			else
			{
				v = -1;
				r = 0;
				r_end = 0;
			}

			while(1)
			{
				while(__any(r_end-r))
				{
					//Vie for control of warp
					if(r_end-r)
					{
						comm[warp_id][0] = lane_id;
					}

					//Winner describes adjlist
					if(comm[warp_id][0] == lane_id)
					{
						comm[warp_id][1] = r;
						comm[warp_id][2] = r_end;
						comm[warp_id][3] = v;
						r = 0; //Same thread cannot win twice
						r_end = 0;
					}

					//Strip mine winner's adjlist
					int r_gather = comm[warp_id][1] + lane_id;
					int r_gather_end = comm[warp_id][2];
					int v_new = comm[warp_id][3];
					while(r_gather < r_gather_end)
					{
						volatile int w = C[r_gather];
						//Assuming no duplicate/self-edges in the graph, no atomics needed
						if(d_row[w] == INT_MAX)
						{
							d_row[w] = d_row[v_new]+1;
							int t = atomicAdd(&Q2_len,1);
							Q2_row[t] = w;
						}
						r_gather += WARP_SIZE;
					}
				}

				k+=blockDim.x;
				if(k < Q_len)
				{
					v = Q_row[k];
					r = R[v];
					r_end = R[v+1];
				}
				else
				{
					v = -1;
					r = 0;
					r_end = 0;
				}

				if((k-threadIdx.x) >= Q_len) //If thread 0 doesn't have work, the entire warp is done
				{
					break;
				}
			}
			__syncthreads();
					
			if(Q2_len == 0)
			{
				break;
			}
			else
			{
				for(int kk=threadIdx.x; kk<Q2_len; kk+=blockDim.x)
				{
					Q_row[kk] = Q2_row[kk];
				}
				__syncthreads();

				if(j==0)
				{
					Q_len = Q2_len;
					Q2_len = 0;
				}
				__syncthreads();
			}
		}
	}
}
