#include "transitive_closure.cuh"

//TODO: Experiment with storage here...get (6GB - overhead) worth of data from one run, store on host, and repeat until the host fills.
void transitive_closure_setup(const device_graph &g, int start, int end, std::vector< std::vector<int> > &d)
{
	//For now, use "standard" grid/block sizes. These can be tuned later on.
	dim3 dimGrid, dimBlock;
        //Returns number of source vertices to store for verification purposes
        size_t sources_to_store = configure_grid(dimGrid,dimBlock,start,end);

	//Device pointers
	int *d_d, *Q_d, *Q2_d;
	pitch p;
	hipEvent_t start_event, end_event;

	//Allocate algorithm-specific memory
	start_clock(start_event,end_event);
	checkCudaErrors(hipMallocPitch((void**)&d_d,&p.d,sizeof(int)*g.n,sources_to_store));
	checkCudaErrors(hipMallocPitch((void**)&Q_d,&p.Q,sizeof(int)*g.n,dimGrid.x));
	checkCudaErrors(hipMallocPitch((void**)&Q2_d,&p.Q2,sizeof(int)*g.n,dimGrid.x));

        size_t GPU_memory_requirement = sizeof(int)*g.n*sources_to_store + 2*sizeof(int)*g.n*dimGrid.x + sizeof(int)*(g.n+1) + sizeof(int)*(g.m); 
        std::cout << "Transitive closure memory requirement: " << GPU_memory_requirement/(1 << 20) << " MB" << std::endl;

	transitive_closure<<<dimGrid,dimBlock>>>(thrust::raw_pointer_cast(g.R.data()),thrust::raw_pointer_cast(g.C.data()),g.n,d_d,Q_d,Q2_d,p,start,end);
	checkCudaErrors(hipPeekAtLastError());

        transfer_result(g,d_d,p.d,sources_to_store,d);

	//Free algorithm-specific memory
	checkCudaErrors(hipFree(Q2_d));
	checkCudaErrors(hipFree(Q_d));
	checkCudaErrors(hipFree(d_d));
	float time = end_clock(start_event,end_event);

	std::cout << "Time for shuffle-based transitive closure: " << std::setprecision(9) << time << " s" << std::endl;
}

